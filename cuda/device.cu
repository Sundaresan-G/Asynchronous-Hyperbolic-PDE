#include <stdlib.h>
#include <unistd.h>
#include <vector>
#include <iostream>
#include <string>
#include "hip/hip_runtime.h"

#include <omp.h>
#include <sched.h>

using namespace std;

#define INFY 2000000000

#define TPB 256

int THREADS_PER_BLOCK = TPB;

extern "C" double* cuda_initialize(double **u_res, int nx, double *device_u_res){
    double *host_u_res = new double[3*(nx+2)];
    for(int i=0; i<2; i++){
        for(int j=0; j<nx+2; j++){
            host_u_res[i*(nx+2)+j] = u_res[i][j];
        }
    }    

    hipMalloc((void **)&device_u_res, 3*(nx+2)*sizeof(double)); 

    // Copy from host to device localPointArray
    hipMemcpy(device_u_res, host_u_res, \
     2*(nx+2)*sizeof(double), hipMemcpyHostToDevice);

    delete host_u_res;

    return device_u_res;

}

extern "C" void setDeviceProps(int rank, int size){
    int nDevices;
    hipGetDeviceCount(&nDevices);
    int device = rank % nDevices;
    hipSetDevice(device);
}

extern "C" void getDeviceProps(int rank, int size){

    int device;

    hipGetDevice(&device);

    int nDevices;
    hipGetDeviceCount(&nDevices);

    char hostname[1024];
    hostname[1023] = '\0';
    gethostname(hostname, 1023);

    printf("My GPU device ID is: %d out of GPU devices: %d in host: %s for MPI rank : %d out of size: %d\n", device, nDevices, hostname, rank, size);

}

__global__ void cuda_compute_next_step(int nx, double cfl, double *device_u_res){

    const int idx = blockIdx.x*blockDim.x + threadIdx.x;

    if(idx>=nx){
        return;
    }

    device_u_res[2*(nx+2)+(idx+1)] = 2*device_u_res[1*(nx+2)+(idx+1)] -\
    device_u_res[0*(nx+2)+(idx+1)] + (cfl*cfl)*(device_u_res[1*(nx+2)+(idx)] -\
     2*device_u_res[1*(nx+2)+(idx+1)] + device_u_res[1*(nx+2)+(idx+2)]);
    
}

extern "C" void cuda_update_first_and_last(double **u_res, int nx, int pos, double *device_u_res){
    double first[2] = {u_res[pos][0], u_res[pos][nx+1]};
    hipMemcpy(&device_u_res[pos*(nx+2)], &first[0], \
     sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(&device_u_res[pos*(nx+2)+nx+1], &first[1], \
     sizeof(double), hipMemcpyHostToDevice);
}

extern "C" void cuda_get_first_last(double **u_res, int nx, double *device_u_res){
    hipMemcpy(&u_res[1][1], &device_u_res[1*(nx+2)+1] , \
        sizeof(double), hipMemcpyDeviceToHost);
    
    hipMemcpy(&u_res[1][nx], &device_u_res[1*(nx+2)+nx] , \
        sizeof(double), hipMemcpyDeviceToHost);
}

__global__ void cuda_kernel_shift(int nx, double *device_u_res){

    const int idx = blockIdx.x*blockDim.x + threadIdx.x;

    if(idx>=nx+2){
        return;
    }

    device_u_res[0*(nx+2)+(idx)] = device_u_res[1*(nx+2)+(idx)];
    
    device_u_res[1*(nx+2)+(idx)] = device_u_res[2*(nx+2)+(idx)];
}

extern "C" void cuda_back_to_host(double **u_res, int last_pos, int nx, double *device_u_res){

    double *host_u_res = new double[3*(nx+2)]; 
    
    hipMemcpy(host_u_res, device_u_res, \
     3*(nx+2)*sizeof(double), hipMemcpyDeviceToHost);

    for(int i=0; i<3; i++){
        for(int j=0; j<nx+2; j++){
            u_res[i][j] = host_u_res[i*(nx+2)+j];
        }
    }  
}

extern "C" void cuda_shift_values(double **u_res, int nx, double *device_u_res){
    int num_blocks = (nx+2 + THREADS_PER_BLOCK - 1)/THREADS_PER_BLOCK;
    
    cuda_kernel_shift<<<num_blocks, THREADS_PER_BLOCK>>>(nx, device_u_res);
}



extern "C" void cuda_compute_values_next_time_step(double **u_res, int nx, double cfl, double *device_u_res){
    int num_blocks = (nx + THREADS_PER_BLOCK - 1)/THREADS_PER_BLOCK;
    
    cuda_compute_next_step<<<num_blocks, THREADS_PER_BLOCK>>>(nx, cfl, device_u_res);
}


extern "C" void cudaDeInitialize(double *device_u_res){

    hipFree(device_u_res);

}
